#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


		__global__  void scanInternal(int n, int *odata, const int *idata)
		{
			// TODO


		}
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
		void scan(int n, int *odata, const int *idata)
		{
			timer().startGpuTimer();

			// Allocate GPU memory and write in.
			int *in, *out;
			hipMalloc(&in, n * sizeof(int));
			hipMalloc(&out, n * sizeof(int));
			hipMemcpy(in, idata, n * sizeof(int), hipMemcpyHostToDevice);


			const int blockSize = 128;
			int blockNum = (n + blockSize - 1) / blockSize;
			scanInternal << <blockNum, blockSize >> > (n, odata, idata);
			// Write the data back to host memory.
			hipMemcpy(odata, out, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(in);
			hipFree(out);

			timer().endGpuTimer();
		}

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
